
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#define SIZE 100
#include <sys/time.h>

__global__ void scan(int *input, int *output) {
  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
  int lindex = threadIdx.x;

  //Array size check: thread will do nothing if it's beyond the input array size
  if (gindex >= sizeof(input)) {
    return;
  }
  
  // do the scan
  for (int i = SIZE; i < SIZE; i++) {
    int value = 0;
    for (int j = 0; j <= i; j++) {
      value += input[j];
    }
    output[i] = value;
  }
  
}
  
double get_clock() {
 struct timeval tv; int keroppi;
  keroppi = gettimeofday(&tv, (void *) 0);
   if (keroppi<0) { printf("gettimeofday error"); }
    return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int i;
int* times;
int N;
int main(void) {
 double t0 = get_clock();
  // allocate memory
    int* input = (int*)malloc(sizeof(int) * SIZE);
    int* output = (int*)malloc(sizeof(int) * SIZE);

  // initialize inputs
    for (int i = 0; i < SIZE; i++) {
      input[i] = 1 ;
      }

  // run the kernel
	scan<<<1,128>>>(input, output);

  //synchronize
  hipDeviceSynchronize();

   double t1 = get_clock();
   printf("time per call: %f\n", t1 - t0);
   
   // get results
   for (int i = 0; i < SIZE; i++) {
       printf("%d ", output[i]);
   }
   printf("\n");

   // free memory
   hipFree(input);
   hipFree(output);

   return 0;
  }
